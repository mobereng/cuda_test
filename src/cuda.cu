#include "cuda.cuh"


SparseMatrixXdRowMajor to_rowmajor(
  const SparseMatrixXd & A
) {
  SparseMatrixXdRowMajor bridge(A.rows(), A.rows());
  VectorTriplet triplets;
  for (int k = 0; k < A.outerSize(); ++k)
    for (SparseMatrixXd::InnerIterator it(A, k); it; ++it)
      triplets.push_back(TripletDouble((int)it.row(), (int)it.col(), it.value()));
  bridge.setFromTriplets(triplets.begin(), triplets.end());
  return bridge;
}


namespace amg {
  MatrixXd gpu(
    const SparseMatrixXd & A_,
    const MatrixXd & b
  ) {
    typedef amgcl::backend::cuda<double> Backend;
    typedef amgcl::make_solver<
            amgcl::amg<
              Backend,
              amgcl::coarsening::smoothed_aggregation,
              amgcl::relaxation::ilu0
            >,
            amgcl::solver::bicgstab<Backend>
            > Solver;

    // Show the name of the GPU we are using:
    int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    std::cout << prop.name << std::endl;

    // Parse matrix to row major
    SparseMatrixXdRowMajor A = to_rowmajor(A_);

    // Prepare params
    Solver::params solver_params;
    solver_params.solver.tol = 1.e-12;
    Backend::params backend_params;
    hipsparseCreate(&backend_params.cusparse_handle);
    Solver solve(A, solver_params, backend_params);

    // Getting solutions
    const double * bx = b.col(0).data();
    const double * by = b.col(1).data();
    const double * bz = b.col(2).data();
    thrust::device_vector<double> dbx(bx, bx + b.rows());
    thrust::device_vector<double> dby(by, by + b.rows());
    thrust::device_vector<double> dbz(bz, bz + b.rows());
    thrust::device_vector<double> dx(A.rows(), 0.0);
    thrust::device_vector<double> dy(A.rows(), 0.0);
    thrust::device_vector<double> dz(A.rows(), 0.0);
    solve(dbx, dx);
    solve(dby, dy);
    solve(dbz, dz);

    // Parsing solution
    //MatrixXd sol(b.rows(), b.cols());
    //sol.col(0) = x;
    //sol.col(1) = y;
    //sol.col(2) = z;
    //return sol;
    return b;
  }
}
